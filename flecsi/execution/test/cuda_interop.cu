#include "hip/hip_runtime.h"
/*~-------------------------------------------------------------------------~~*
 * Copyright (c) 2019 Triad National Security, LLC
 * All rights reserved.
 *~-------------------------------------------------------------------------~~*/

//#include <cinchtest.h>
#include <assert.h>
#include <iostream>
//#include <flecsi/execution/context.h>
//#include <flecsi/execution/execution.h>

//clog_register_tag(devel_handle);

namespace flecsi {
namespace execution {

//cuda function to be tested.  calculates y=a*x where:
// a is a floating point scalar,
// x and y are device pointers to floating point arrays of equal size
__global__ void axpy(float a, float* x, float* y) {
  y[threadIdx.x] = a * x[threadIdx.x];
}

//Just launch the kernel.  Assumes x & y are existing device arrays of length len
void simpleKernelLaunch(float a, float* x, float* y, int len){
	axpy<<<1,len>>>(a, x, y);
}

// driver for test.  Prints out some information about the cuda environment,
// generates input data for the test, calls the test, then transfers data 
// back to host.  
void runCuda() {
  int runtime_ver;
  hipRuntimeGetVersion(&runtime_ver);
  std::cout << "CUDA Runtime: " << runtime_ver << std::endl;

  int driver_ver;
  hipDriverGetVersion(&driver_ver);
  std::cout << "CUDA Driver: " << driver_ver << std::endl;

  const int kDataLen = 4;

  float a = 2.0f;
  float host_x[kDataLen] = {1.0f, 2.0f, 3.0f, 4.0f};
  float host_y[kDataLen];

  // Copy input data to device.
  float* device_x;
  float* device_y;
  hipMalloc(&device_x, kDataLen * sizeof(float));
  hipMalloc(&device_y, kDataLen * sizeof(float));
  hipMemcpy(device_x, host_x, kDataLen * sizeof(float), hipMemcpyHostToDevice);

  // Launch the kernel.
  simpleKernelLaunch(a, device_x, device_y, kDataLen);
  //axpy<<<1, kDataLen>>>(a, device_x, device_y);

  // Copy output data to host.
  hipDeviceSynchronize();
  hipMemcpy(host_y, device_y, kDataLen * sizeof(float), hipMemcpyDeviceToHost);

  // Print the results.
  for (int i = 0; i < kDataLen; ++i) {
    std::cout << "y[" << i << "] = " << host_y[i] << std::endl;
    assert(host_y[i] == a * host_x[i]);
  }
  return;
}

}} //end namespace

main(){
	flecsi::execution::runCuda();
}
